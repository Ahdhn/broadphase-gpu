#include "hip/hip_runtime.h"

#include <gpubf/aabb.cuh>
#include <gpubf/util.cuh>

#include <spdlog/spdlog.h>

using namespace stq::gpu;

void setup(int devId, int &smemSize, int &threads, int &nbox) {
  // Host code
  // int maxbytes = 98304; // 96 KB
  // hipFuncSetAttribute(reinterpret_cast<const void*>(get_collision_pairs),
  // hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

  int maxSmem;
  hipDeviceGetAttribute(&maxSmem, hipDeviceAttributeMaxSharedMemoryPerBlock, devId);
  spdlog::trace("Max shared Memory per Block: {:d} B", maxSmem);

  int maxThreads;
  hipDeviceGetAttribute(&maxThreads, hipDeviceAttributeMaxThreadsPerBlock, devId);
  spdlog::trace("Max threads per Block: {:d} thrds", maxThreads);

  nbox = nbox ? nbox : std::max((int)(maxSmem / sizeof(Aabb)) / maxThreads, 1);
  spdlog::trace("Boxes per Thread: {:d}", nbox);

  // divide threads by an arbitrary number as long as its reasonable >64
  if (!threads) {
    hipDeviceGetAttribute(&threads, hipDeviceAttributeMaxThreadsPerMultiProcessor,
                           devId);

    spdlog::trace("Max threads per Multiprocessor: {:d} thrds", threads);
  }
  smemSize = nbox * threads * sizeof(Aabb);

  while (smemSize > maxSmem || threads > maxThreads) {
    threads--;
    smemSize = nbox * threads * sizeof(Aabb);
  }
  spdlog::trace("Actual threads per Block: {:d} thrds", threads);
  spdlog::trace("Shared mem alloc: {:d} B", smemSize);

  // int warpSize;
  // hipDeviceGetAttribute(&warpSize,
  //     hipDeviceAttributeWarpSize, devId);
  // spdlog::trace("Warp Size: {:d}", warpSize);

  // bank conflict avoid
  // hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

  // hipSharedMemConfig bankSize;
  // hipDeviceGetSharedMemConfig(&bankSize);
  // spdlog::trace("Bank size: {:d}", bankSize );

  return;
}