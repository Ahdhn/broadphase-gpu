#include "hip/hip_runtime.h"

#include <gpubf/aabb.cuh>
#include <gpubf/util.cuh>

#include <spdlog/spdlog.h>

using namespace ccdgpu;
using namespace std;

void setup(int devId, int &smemSize, int &threads, int &nbox) {
  // Host code
  // int maxbytes = 98304; // 96 KB
  // hipFuncSetAttribute(reinterpret_cast<const void*>(get_collision_pairs),
  // hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

  int maxSmem;
  hipDeviceGetAttribute(&maxSmem, hipDeviceAttributeMaxSharedMemoryPerBlock, devId);
  spdlog::trace("Max shared Memory per Block: {:i} B", maxSmem);

  int maxThreads;
  hipDeviceGetAttribute(&maxThreads, hipDeviceAttributeMaxThreadsPerBlock, devId);
  spdlog::trace("Max threads per Block: {:i} thrds", maxThreads);

  nbox = nbox ? nbox : std::max((int)(maxSmem / sizeof(Aabb)) / maxThreads, 1);
  spdlog::trace("Boxes per Thread: {:i}", nbox);

  // divide threads by an arbitrary number as long as its reasonable >64
  if (!threads) {
    hipDeviceGetAttribute(&threads, hipDeviceAttributeMaxThreadsPerMultiProcessor,
                           devId);

    spdlog::trace("Max threads per Multiprocessor: {:i} thrds", threads);
  }
  smemSize = nbox * threads * sizeof(Aabb);

  while (smemSize > maxSmem || threads > maxThreads) {
    threads--;
    smemSize = nbox * threads * sizeof(Aabb);
  }
  spdlog::trace("Actual threads per Block: {:i} thrds", threads);
  spdlog::trace("Shared mem alloc: {:i} B", smemSize);

  // int warpSize;
  // hipDeviceGetAttribute(&warpSize,
  //     hipDeviceAttributeWarpSize, devId);
  // spdlog::trace("Warp Size: {:i}", warpSize);

  // bank conflict avoid
  // hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

  // hipSharedMemConfig bankSize;
  // hipDeviceGetSharedMemConfig(&bankSize);
  // spdlog::trace("Bank size: {:i}", bankSize );

  return;
}