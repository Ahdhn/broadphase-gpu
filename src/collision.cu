#include "hip/hip_runtime.h"
#include <gpubf/collision.h>

__global__ void count_collisions(Aabb * boxes, int * count, int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;
    
    const Aabb& a = boxes[tid];
    const Aabb& b = boxes[tid];

    bool collides = 
        a.max[0] >= b.min[0] && a.min[0] <= b.max[0] &&
        a.max[1] >= b.min[1] && a.min[1] <= b.max[1] &&
        a.max[2] >= b.min[2] && a.min[2] <= b.max[2];

    if (collides)
        atomicAdd(count, 1);
}

__device__ bool does_collide(const Aabb& x, const Aabb& y)
{
    return 
        x.max[0] >= y.min[0] && x.min[0] <= y.max[0] &&
            x.max[1] >= y.min[1] && x.min[1] <= y.max[1] &&
            x.max[2] >= y.min[2] && x.min[2] <= y.max[2];
}

__device__ void check_add_overlap(bool collides, const Aabb& x, const Aabb& y, int * count, int * overlaps, int G)
{
    if (collides)
        {
            int i = atomicAdd(count, 1);

            if (2*i + 1 < G)
            {
                overlaps[2*i] = x.id;
                overlaps[2*i+1] = y.id;
            }
        }

}

__global__ void get_collision_pairs(Aabb * boxes, int * count, int * overlaps, int N, int G, const int nBoxesPerThread)
{   
        extern __shared__ Aabb s_objects[];
    
        int threadRowId = nBoxesPerThread*blockIdx.x * blockDim.x + threadIdx.x;
        int threadColId = nBoxesPerThread*blockIdx.y * blockDim.y + threadIdx.y;

        if (threadRowId >= N || threadColId >= N || threadColId >= threadRowId) return;


        for (int shift = 0; shift < nBoxesPerThread; shift++)
        {
            int tidRow = threadRowId + shift*blockDim.x;
            int xIdx = (shift)*BLOCK_SIZE_1D + threadIdx.x;
            s_objects[xIdx] = boxes[tidRow];

            int tidCol = threadColId + shift*blockDim.y;
            int yIdx = (shift+nBoxesPerThread)*BLOCK_SIZE_1D + threadIdx.y;
            s_objects[yIdx] = boxes[tidCol];
        }

        for (int i=0; i < nBoxesPerThread; i++)
        {
            for (int j=nBoxesPerThread; j < 2*nBoxesPerThread; j++)
            {
                const Aabb& x = s_objects[i*BLOCK_SIZE_1D + threadIdx.x];      
                const Aabb& y = s_objects[j*BLOCK_SIZE_1D + threadIdx.y];
                
                bool collides = does_collide(x,y);
                check_add_overlap(collides, x, y, count, overlaps, G);
            }
        }
    
}

__global__ void reset_counter(int * counter){
    *counter = 0;
}

__global__ void get_collision_pairs_old(Aabb * boxes, int * count, int * overlaps, int N, int G)
{
    
        int threadRowId = blockIdx.x * blockDim.x + threadIdx.x;
        int threadColId = blockIdx.y * blockDim.y + threadIdx.y;
       
        if (threadRowId >= N || threadColId >= N || threadColId >= threadRowId) return;
    
        const Aabb& a = boxes[threadRowId];
        const Aabb& b = boxes[threadColId];
        bool collides = does_collide(a,b);
        check_add_overlap(collides, a, b, count, overlaps, G);
}
