#include "hip/hip_runtime.h"
// #include <gpubf/klee.cuh>
#include <gpubf/queue.cuh>
#include <iostream>



// __device__ __host__ Enqueue::Enqueue()
// {
// 	heap_size = 0;
// 	capacity = HEAP_SIZE;
// }


// // Inserts a new key 'k'
// __device__ bool Enqueue::push(const Cell &k)
// { // to avoid overflow, instead of comparing with capacity, we compare with capacity -1
// 	if (heap_size == capacity - 1)
// 	{
// 		return false;
// 	}

// 	// First insert the new key at the end

// 	int i = heap_size;

// 	harr[i] = k;

// 	heap_size++;

// 	return true;
// }

// // Method to remove minimum element (or root) from min heap
// __device__ Cell Enqueue::pop()
// {

// 	Cell root = harr[0];

// 	harr[0] = harr[heap_size - 1];
// 	heap_size--;

// 	MinHeapify();

// 	return root;
// }

// __device__ void Enqueue::MinHeapify()
// {
// 	int itr = 0;

// 	while (itr != heap_size)
//     {
// 			swap(harr[itr], harr[itr+1]);
//             itr++;
//     }
// }

// __device__ bool Enqueue::empty()
// {
// 	return (heap_size == 0);
// }

// __device__ int Enqueue::size()
// {
// 	return heap_size;
// }

// // A utility function to swap two elements
// __device__ void swap(Cell& x, Cell& y)
// {
// 	Cell temp;
// 	temp = x;
// 	x = y;
// 	y = temp;
// }

__device__ __host__ Queue::Queue()
{
	heap_size = 0;
	capacity = HEAP_SIZE;
}

__device__ int2 Queue::pop()
{
	int itr = 0;
	while (harr[itr].x < 0)
		itr = (itr + 1) % heap_size;
	int2 val = harr[itr];
	harr[itr].x = -1; //set as removed
	return val;
}

__device__ void Queue::push(int2 pair)
{
	printf("%i %i\n", heap_size, capacity);
	if (heap_size < capacity)
	{
		harr[heap_size] = pair;
		heap_size++;
		printf("Pushed pair (%i,%i) to harr[%i] with cap %i\n", pair.x, pair.y, heap_size, capacity);
		return;
	}
	
	int itr = 0;

	while (harr[itr].x > 0)
		itr = (itr + 1) % heap_size;
	harr[itr] = pair;
	printf("Pushed pair (%i,%i) to harr[%i]\n", pair.x, pair.y, itr);
	return;
	
}

__device__ int Queue::size()
{
	return heap_size;
}