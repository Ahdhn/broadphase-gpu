#include "hip/hip_runtime.h"
#include <gpubf/aabb.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

// __host__ __device__ half3 make_half3(__half x, __half y, __half z) {
//   half3 t;
//   t.x = x;
//   t.y = y;
//   t.z = z;
//   return t;
// }

// __host__ __device__ half3 make_half3(float x, float y, float z) {
//   half3 t;
//   t.x = __float2half(x);
//   t.y = __float2half(y);
//   t.z = __float2half(z);
//   return t;
// }

namespace ccdgpu {

#pragma omp declare reduction (merge : std::vector<Aabb> : omp_out.insert(omp_out.end(), omp_in.begin(), omp_in.end()))
#define run_threads std::min(omp_get_max_threads(), 64)

#ifdef CCD_USE_DOUBLE
#warning Using Double
__host__ __device__ Scalar3 make_Scalar3(const Scalar a, const Scalar b,
                                         const Scalar &c) {
  return make_double3(a, b, c);
}
__host__ __device__ Scalar2 make_Scalar2(const Scalar a, const Scalar b) {
  return make_double2(a, b);
}
#else
__host__ __device__ Scalar3 make_Scalar3(const Scalar a, const Scalar b,
                                         const Scalar &c) {
  return make_float3(a, b, c);
}
__host__ __device__ Scalar2 make_Scalar2(const Scalar a, const Scalar b) {
  return make_float2(a, b);
}
#warning Using Float
#endif

__host__ __device__ bool is_face(const Aabb &x) { return x.vertexIds.z >= 0; };

__host__ __device__ bool is_face(const int3 &vids) { return vids.z >= 0; };

__host__ __device__ bool is_edge(const Aabb &x) {
  return x.vertexIds.z < 0 && x.vertexIds.y >= 0;
};

__host__ __device__ bool is_edge(const int3 &vids) {
  return vids.z < 0 && vids.y >= 0;
};

__host__ __device__ bool is_vertex(const Aabb &x) {
  return x.vertexIds.z < 0 && x.vertexIds.y < 0;
};

__host__ __device__ bool is_vertex(const int3 &vids) {
  return vids.z < 0 && vids.y < 0;
};

__host__ __device__ bool is_valid_pair(const Aabb &a, const Aabb &b) {
  return (is_vertex(a) && is_face(b)) || (is_face(a) && is_vertex(b)) ||
         (is_edge(a) && is_edge(b));
};

__host__ __device__ bool is_valid_pair(const int3 &a, const int3 &b) {
  return (is_vertex(a) && is_face(b)) || (is_face(a) && is_vertex(b)) ||
         (is_edge(a) && is_edge(b));
};

float nextafter_up(float x) { return nextafterf(x, x + 1.); };
float nextafter_down(float x) { return nextafterf(x, x - 1.); };

void addEdges(const Eigen::MatrixXd &vertices_t0,
              const Eigen::MatrixXd &vertices_t1, const Eigen::MatrixXi &edges,
              vector<Aabb> &boxes) {
#pragma omp parallel for num_threads(run_threads), reduction(merge : boxes)
  for (unsigned long i = 0; i < edges.rows(); i++) {
    Eigen::MatrixXd edge_vertex0_t0 = vertices_t0.row(edges(i, 0));
    Eigen::MatrixXd edge_vertex1_t0 = vertices_t0.row(edges(i, 1));
    Eigen::MatrixXd edge_vertex0_t1 = vertices_t1.row(edges(i, 0));
    Eigen::MatrixXd edge_vertex1_t1 = vertices_t1.row(edges(i, 1));

    Eigen::MatrixXd points(4, edge_vertex0_t0.size());
    points.row(0) = edge_vertex0_t0;
    points.row(1) = edge_vertex1_t0;
    points.row(2) = edge_vertex0_t1;
    points.row(3) = edge_vertex1_t1;

    int vertexIds[3] = {edges(i, 0), edges(i, 1), -edges(i, 0) - 1};
#ifdef CCD_USE_DOUBLE
    Eigen::Vector3d lower_bound = points.colwise().minCoeff();
    Eigen::Vector3d upper_bound = points.colwise().maxCoeff();
#else

    Eigen::MatrixXf lower_bound =
        points.colwise().minCoeff().unaryExpr(&nextafter_down);
    Eigen::MatrixXf upper_bound =
        points.colwise().maxCoeff().unaryExpr(&nextafter_up);
#endif
    boxes.emplace_back(boxes.size(), i, vertexIds, lower_bound.array().data(),
                       upper_bound.array().data());
  }
}

void addVertices(const Eigen::MatrixXd &vertices_t0,
                 const Eigen::MatrixXd &vertices_t1, vector<Aabb> &boxes) {
#pragma omp parallel for num_threads(run_threads), reduction(merge : boxes)
  for (unsigned long i = 0; i < vertices_t0.rows(); i++) {
    Eigen::MatrixXd vertex_t0 = vertices_t0.row(i);
    Eigen::MatrixXd vertex_t1 = vertices_t1.row(i);

    Eigen::MatrixXd points(2, vertex_t0.size());
    points.row(0) = vertex_t0;
    points.row(1) = vertex_t1;

    int vertexIds[3] = {i, -i - 1, -i - 1};

#ifdef CCD_USE_DOUBLE
    Eigen::MatrixXd lower_bound = points.colwise().minCoeff();
    Eigen::MatrixXd upper_bound = points.colwise().maxCoeff();
#else

    Eigen::MatrixXf lower_bound =
        points.colwise().minCoeff().unaryExpr(&nextafter_down);
    Eigen::MatrixXf upper_bound =
        points.colwise().maxCoeff().unaryExpr(&nextafter_up);
#endif
    boxes.emplace_back(boxes.size(), i, vertexIds, lower_bound.array().data(),
                       upper_bound.array().data());
  }
}

void addFaces(const Eigen::MatrixXd &vertices_t0,
              const Eigen::MatrixXd &vertices_t1, const Eigen::MatrixXi &faces,
              vector<Aabb> &boxes) {
#pragma omp parallel for num_threads(run_threads), reduction(merge : boxes)
  for (unsigned long i = 0; i < faces.rows(); i++) {
    Eigen::MatrixXd face_vertex0_t0 = vertices_t0.row(faces(i, 0));
    Eigen::MatrixXd face_vertex1_t0 = vertices_t0.row(faces(i, 1));
    Eigen::MatrixXd face_vertex2_t0 = vertices_t0.row(faces(i, 2));
    Eigen::MatrixXd face_vertex0_t1 = vertices_t1.row(faces(i, 0));
    Eigen::MatrixXd face_vertex1_t1 = vertices_t1.row(faces(i, 1));
    Eigen::MatrixXd face_vertex2_t1 = vertices_t1.row(faces(i, 2));

    Eigen::MatrixXd points(6, face_vertex0_t0.size());
    points.row(0) = face_vertex0_t0;
    points.row(1) = face_vertex1_t0;
    points.row(2) = face_vertex2_t0;
    points.row(3) = face_vertex0_t1;
    points.row(4) = face_vertex1_t1;
    points.row(5) = face_vertex2_t1;

    int vertexIds[3] = {faces(i, 0), faces(i, 1), faces(i, 2)};

#ifdef CCD_USE_DOUBLE
    Eigen::Vector3d lower_bound = points.colwise().minCoeff();
    Eigen::Vector3d upper_bound = points.colwise().maxCoeff();
#else

    Eigen::MatrixXf lower_bound =
        points.colwise().minCoeff().unaryExpr(&nextafter_down);
    Eigen::MatrixXf upper_bound =
        points.colwise().maxCoeff().unaryExpr(&nextafter_up);
#endif
    boxes.emplace_back(boxes.size(), i, vertexIds, lower_bound.array().data(),
                       upper_bound.array().data());
  }
};

} // namespace ccdgpu