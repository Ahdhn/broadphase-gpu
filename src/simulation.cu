#include "hip/hip_runtime.h"
#include <gpubf/simulation.h>

void run_simulation(Aabb* boxes, int N) {
    // int N = 200000;
    // Aabb boxes[N];
    // for (int i = 0; i<N; i++)
    // {
    //     boxes[i] = Aabb(i);
    //     // printf("box %i created\n", boxes[i].id);
    // }

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    reset_counter<<<1,1>>>(d_counter);
    hipDeviceSynchronize();

     int collisions;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    // int bytes_mem_intrfce = 352 >> 3;
    // int mem_clock_rate = 1376 << 1;
    // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10, 3);

    // Set up timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get number of collisions
    hipEventRecord(start);
    count_collisions<<<1,1>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("(count_collisions<<<1,1>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    printf("Effective Bandwidth (GB/s): %.6f (GB/s)\n", 32*2/milliseconds/1e6);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<1,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<1,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<2,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<2,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<56,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<56,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<256,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<256,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    return;
    // printf("%zu\n", sizeof(Aabb));


    // Retrieve count from GPU and print out
    // int counter;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    // printf("count: %d\n", counter);
    // return 0;
}