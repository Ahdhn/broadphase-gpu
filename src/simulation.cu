#include "hip/hip_runtime.h"
#include <gpubf/simulation.h>



int setup_shared_memory()
{
    // Host code
    int maxbytes = 98304; // 96 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(get_collision_pairs), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

    int smemSize;
    int devId = 0;
    hipDeviceGetAttribute(&smemSize, 
        hipDeviceAttributeSharedMemPerBlockOptin, devId);
    printf("Shared Memory per Block: %i B\n", smemSize);
    
    int warpSize;
    hipDeviceGetAttribute(&warpSize, 
        hipDeviceAttributeWarpSize, devId);
    printf("Warp Size: %i\n", warpSize);
    
    // bank conflict avoid
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    hipSharedMemConfig bankSize;
    hipDeviceGetSharedMemConfig(&bankSize);
    printf("Bank size: %i\n", bankSize );
    

    return smemSize;
}


void run_collision_counter(Aabb* boxes, int N) {

    // int N = 200000;
    // Aabb boxes[N];
    // for (int i = 0; i<N; i++)
    // {
    //     boxes[i] = Aabb(i);
    //     // printf("box %i created\n", boxes[i].id);
    // }

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    reset_counter<<<1,1>>>(d_counter);
    hipDeviceSynchronize();

     int collisions;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    // int bytes_mem_intrfce = 352 >> 3;
    // int mem_clock_rate = 1376 << 1;
    // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10, 3);

    // Set up timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get number of collisions
    hipEventRecord(start);
    count_collisions<<<1,1>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("(count_collisions<<<1,1>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    printf("Effective Bandwidth (GB/s): %.6f (GB/s)\n", 32*2/milliseconds/1e6);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<1,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<1,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<2,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<2,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<56,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<56,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<256,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<256,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    return;
    // printf("%zu\n", sizeof(Aabb));


    // Retrieve count from GPU and print out
    // int counter;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    // printf("count: %d\n", counter);
    // return 0;
}

void run_scaling(const Aabb* boxes,  int N, int desiredBoxesPerThread, vector<unsigned long>& finOverlaps)
{
    hipSetDevice(1);

    int smemSize = setup_shared_memory();
    const int nBoxesPerThread = desiredBoxesPerThread ? desiredBoxesPerThread : smemSize / sizeof(Aabb) / (2*(BLOCK_PADDED));
    printf("Boxes per Thread: %i\n", nBoxesPerThread);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // guess overlaps size
    int guess = 0;

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    reset_counter<<<1,1>>>(d_count);
    hipDeviceSynchronize();

    int2 * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int2)*(guess));

    dim3 block(BLOCK_SIZE_1D,BLOCK_SIZE_1D);
    // dim3 grid ( (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D,  (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D );
    int grid_dim_1d = (N+BLOCK_SIZE_1D)/ BLOCK_SIZE_1D / nBoxesPerThread;
    dim3 grid( grid_dim_1d, grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));

    printf("Shared mem alloc: %i B\n", nBoxesPerThread*2*(BLOCK_PADDED)*sizeof(Aabb) );
    hipEventRecord(start);
    get_collision_pairs<<<grid, block, 49152>>>(d_boxes, d_count, d_overlaps, N, guess, nBoxesPerThread);
    // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, guess);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // hipDeviceSynchronize();

    int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (count > guess) //we went over
    {
        printf("Running again\n");
        hipFree(d_overlaps);
        hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
        reset_counter<<<1,1>>>(d_count);
        hipDeviceSynchronize();
        hipEventRecord(start);
        get_collision_pairs<<<grid, block, 49152>>>(d_boxes, d_count, d_overlaps, N, count, nBoxesPerThread);
        // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, 2*count);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        // hipDeviceSynchronize();
    }
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Collisions: %i\n", count);
    printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
    printf("Boxes: %i\n", N);
    printf("Elapsed time: %.9f ms/box\n", milliseconds/N);

    int2 * overlaps =  (int2*)malloc(sizeof(int2) * (count));
    hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost);


    hipFree(d_overlaps);
    for (size_t i=0; i< count; i++)
    {
        finOverlaps.push_back(overlaps[i].x);
        finOverlaps.push_back(overlaps[i].y);
        
        // const Aabb& a = boxes[overlaps[2*i]];
        // const Aabb& b = boxes[overlaps[2*i + 1]];
        // if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
        // {
        //     finOverlaps.push_back(a.ref_id);
        //     finOverlaps.push_back(b.ref_id);
        // }
        // else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
        // {
        //     finOverlaps.push_back(b.ref_id);
        //     finOverlaps.push_back(a.ref_id);
        // }
        // else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
        // {
        //     finOverlaps.push_back(b.ref_id);
        //     finOverlaps.push_back(a.ref_id);
        // }
    }

    printf("Total(filt.) overlaps: %lu\n", finOverlaps.size() / 2);
    free(overlaps);
    // free(counter);
    // free(counter);
    hipFree(d_count); 

}