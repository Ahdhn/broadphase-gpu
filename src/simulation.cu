#include "hip/hip_runtime.h"
#include <stq/gpu/simulation.cuh>

#include <stq/gpu/collision.cuh>
#include <stq/gpu/queue.cuh>
#include <stq/gpu/sweep.cuh>
#include <stq/gpu/timer.cuh>
#include <stq/gpu/memory.cuh>

#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <tbb/enumerable_thread_specific.h>
#include <tbb/parallel_for.h>
#include <tbb/global_control.h>

#include <spdlog/spdlog.h>

namespace stq::gpu {

extern MemHandler *memhandle;

void setup(int devId, int &smemSize, int &threads, int &nboxes);

void run_collision_counter(Aabb *boxes, int N) {

  // int N = 200000;
  // Aabb boxes[N];
  // for (int i = 0; i<N; i++)
  // {
  //     boxes[i] = Aabb(i);
  //     // spdlog::trace("box {:d} created", boxes[i].id);
  // }

  // Allocate boxes to GPU
  Aabb *d_boxes;
  hipMalloc((void **)&d_boxes, sizeof(Aabb) * N);
  hipMemcpy(d_boxes, boxes, sizeof(Aabb) * N, hipMemcpyHostToDevice);

  // Allocate counter to GPU + set to 0 collisions
  int *d_counter;
  hipMalloc((void **)&d_counter, sizeof(int));
  reset_counter<<<1, 1>>>(d_counter);
  hipDeviceSynchronize();

  int collisions;
  // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

  // int bytes_mem_intrfce = 352 >> 3;
  // int mem_clock_rate = 1376 << 1;
  // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10,
  // 3);

  // Set up timer
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Get number of collisions
  hipEventRecord(start);
  count_collisions<<<1, 1>>>(d_boxes, d_counter, N);
  hipEventRecord(stop);
  hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  spdlog::trace("(count_collisions<<<1,1>>>)\n");
  spdlog::trace("Elapsed time: {:.6f} ms", milliseconds);
  spdlog::trace("Elapsed time: {:.6f} ms/c", milliseconds / collisions);
  spdlog::trace("Collision: {:d}", collisions);
  spdlog::trace("Effective Bandwidth (GB/s): {:.6f} (GB/s)",
                32 * 2 / milliseconds / 1e6);

  reset_counter<<<1, 1>>>(d_counter);
  hipEventRecord(start);
  count_collisions<<<1, 1024>>>(d_boxes, d_counter, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  spdlog::trace("(count_collisions<<<1,1024>>>)");
  spdlog::trace("Elapsed time: {:.6f} ms", milliseconds);
  spdlog::trace("Elapsed time: {:.6f} ms/c", milliseconds / collisions);
  spdlog::trace("Collision: {:d}", collisions);

  reset_counter<<<1, 1>>>(d_counter);
  hipEventRecord(start);
  count_collisions<<<2, 1024>>>(d_boxes, d_counter, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  spdlog::trace("(count_collisions<<<2,1024>>>)");
  spdlog::trace("Elapsed time: {:.6f} ms", milliseconds);
  spdlog::trace("Elapsed time: {:.6f} ms/c", milliseconds / collisions);
  spdlog::trace("Collision: {:d}", collisions);

  reset_counter<<<1, 1>>>(d_counter);
  hipEventRecord(start);
  count_collisions<<<56, 1024>>>(d_boxes, d_counter, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  spdlog::trace("(count_collisions<<<56,1024>>>)");
  spdlog::trace("Elapsed time: {:.6f} ms", milliseconds);
  spdlog::trace("Elapsed time: {:.9f} ms/c", milliseconds / collisions);
  spdlog::trace("Collision: {:d}", collisions);

  reset_counter<<<1, 1>>>(d_counter);
  hipEventRecord(start);
  count_collisions<<<256, 1024>>>(d_boxes, d_counter, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  spdlog::trace("(count_collisions<<<256,1024>>>)");
  spdlog::trace("Elapsed time: {:.6f} ms", milliseconds);
  spdlog::trace("Elapsed time: {:.9f} ms/c", milliseconds / collisions);
  spdlog::trace("Collision: {:d}", collisions);
  return;
  // spdlog::trace("%zu", sizeof(Aabb));

  // Retrieve count from GPU and print out
  // int counter;
  // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  // spdlog::trace("count: {:d}", counter);
  // return 0;
}

void run_scaling(const Aabb *boxes, int N, int desiredBoxesPerThread,
                 std::vector<unsigned long> &finOverlaps) {

  int devId = 0;
  hipSetDevice(devId);

  int smemSize;
  int threads;

  setup(devId, smemSize, threads, desiredBoxesPerThread);
  const int nBoxesPerThread =
    desiredBoxesPerThread ? desiredBoxesPerThread
                          : smemSize / sizeof(Aabb) / (2 * (BLOCK_PADDED));
  spdlog::trace("Boxes per Thread: {:d}", nBoxesPerThread);

  finOverlaps.clear();
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // guess overlaps size
  int guess = 0;

  // Allocate boxes to GPU
  Aabb *d_boxes;
  hipMalloc((void **)&d_boxes, sizeof(Aabb) * N);
  hipMemcpy(d_boxes, boxes, sizeof(Aabb) * N, hipMemcpyHostToDevice);

  // Allocate counter to GPU + set to 0 collisions
  int *d_count;
  hipMalloc((void **)&d_count, sizeof(int));
  reset_counter<<<1, 1>>>(d_count);
  hipDeviceSynchronize();

  // Count collisions
  count_collisions<<<1, 1>>>(d_boxes, d_count, N);
  int count;
  hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
  reset_counter<<<1, 1>>>(d_count);
  spdlog::trace("Total collisions from counting: {:d}", count);

  int2 *d_overlaps;
  hipMalloc((void **)&d_overlaps, sizeof(int2) * (guess));

  dim3 block(BLOCK_SIZE_1D, BLOCK_SIZE_1D);
  // dim3 grid ( (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D,
  // (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D );
  int grid_dim_1d = (N + BLOCK_SIZE_1D) / BLOCK_SIZE_1D / nBoxesPerThread;
  dim3 grid(grid_dim_1d, grid_dim_1d);
  spdlog::trace("Grid dim (1D): {:d}", grid_dim_1d);
  spdlog::trace("Box size: {:d}", sizeof(Aabb));

  long long *d_queries;
  hipMalloc((void **)&d_queries, sizeof(long long) * (1));
  reset_counter<<<1, 1>>>(d_queries);

  spdlog::trace("Shared mem alloc: {:d} B",
                nBoxesPerThread * 2 * (BLOCK_PADDED) * sizeof(Aabb));
  hipEventRecord(start);
  get_collision_pairs<<<grid, block,
                        nBoxesPerThread * 2 * (BLOCK_PADDED) * sizeof(Aabb)>>>(
    d_boxes, d_count, d_overlaps, N, guess, nBoxesPerThread, d_queries);
  // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N,
  // guess);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  // hipDeviceSynchronize();

  long long queries;
  hipMemcpy(&queries, d_queries, sizeof(long long), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  spdlog::trace("queries: {:d}", queries);
  spdlog::trace("needed queries: {:d}", (long long)N * (N - 1) / 2);

  // int count;
  hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  if (count > guess) // we went over
  {
    spdlog::trace("Running again\n");
    hipFree(d_overlaps);
    hipMalloc((void **)&d_overlaps, sizeof(int2) * (count));
    reset_counter<<<1, 1>>>(d_count);
    hipDeviceSynchronize();
    hipEventRecord(start);
    get_collision_pairs<<<
      grid, block, nBoxesPerThread * 2 * (BLOCK_PADDED) * sizeof(Aabb)>>>(
      d_boxes, d_count, d_overlaps, N, count, nBoxesPerThread, d_queries);
    // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N,
    // 2*count);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // hipDeviceSynchronize();
  }

  spdlog::trace("Elapsed time: {:.6f} ms", milliseconds);
  spdlog::trace("Collisions: {:d}", count);
  spdlog::trace("Elapsed time: {:.9f} ms/collision", milliseconds / count);
  spdlog::trace("Boxes: {:d}", N);
  spdlog::trace("Elapsed time: {:.9f} ms/box", milliseconds / N);
  // spdlog::trace("Elapsed time: {:.15f} us/query", (milliseconds*1000)/((long
  // long)N*N/2));

  int2 *overlaps = (int2 *)malloc(sizeof(int2) * (count));
  gpuErrchk(hipMemcpy(overlaps, d_overlaps, sizeof(int2) * (count),
                       hipMemcpyDeviceToHost));

  hipFree(d_overlaps);
  // for (size_t i=0; i< count; i++)
  // {
  //     // finOverlaps.push_back(overlaps[i].x, overlaps[i].y);
  //     // finOverlaps.push_back(overlaps[i].y);

  //     const Aabb& a = boxes[overlaps[i].x];
  //     const Aabb& b = boxes[overlaps[i].y];
  //     if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
  //     {
  //         finOverlaps.push_back(a.ref_id);
  //         finOverlaps.push_back(b.ref_id);
  //     }
  //     else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
  //     {
  //         finOverlaps.push_back(b.ref_id);
  //         finOverlaps.push_back(a.ref_id);
  //     }
  //     else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
  //     {
  //         finOverlaps.push_back(min(a.ref_id, b.ref_id));
  //         finOverlaps.push_back(max(a.ref_id, b.ref_id));
  //     }
  // }

  spdlog::trace("Total(filt.) overlaps: {:d}", finOverlaps.size() / 2);
  free(overlaps);
  // free(counter);
  // free(counter);
  hipFree(d_count);
  hipDeviceReset();
}

struct sorter {};

struct sort_aabb_x : sorter {
  __device__ bool operator()(const Aabb &a, const Aabb &b) const {
    return (a.min.x < b.min.x);
  }

  __device__ bool operator()(const Scalar3 &a, const Scalar3 &b) const {
    return (a.x < b.x);
  }

  __device__ bool operator()(const Scalar2 &a, const Scalar2 &b) const {
    return (a.x < b.x);
  }

  __device__ bool operator()(const RankBox &a, const RankBox &b) const {
    return (a.aabb->min.x < b.aabb->min.x);
  }
};

typedef tbb::enumerable_thread_specific<std::vector<std::pair<int, int>>>
  ThreadSpecificOverlaps;

void merge_local_overlaps(const ThreadSpecificOverlaps &storages,
                          std::vector<std::pair<int, int>> &overlaps) {
  overlaps.clear();
  size_t num_overlaps = overlaps.size();
  for (const auto &local_overlaps : storages) {
    num_overlaps += local_overlaps.size();
  }
  // serial merge!
  overlaps.reserve(num_overlaps);
  for (const auto &local_overlaps : storages) {
    overlaps.insert(overlaps.end(), local_overlaps.begin(),
                    local_overlaps.end());
  }
}

void run_sweep_multigpu(const Aabb *boxes, int N, int nbox,
                        std::vector<std::pair<int, int>> &finOverlaps,
                        int &threads, int &devcount) {
  spdlog::critical("default threads {}", tbb::info::default_concurrency());
  ThreadSpecificOverlaps storages;

  float milliseconds = 0;
  int device_init_id = 0;

  int smemSize;
  setup(device_init_id, smemSize, threads, nbox);

  hipSetDevice(device_init_id);

  finOverlaps.clear();
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate boxes to GPU
  Aabb *d_boxes;
  hipMalloc((void **)&d_boxes, sizeof(Aabb) * N);
  hipMemcpy(d_boxes, boxes, sizeof(Aabb) * N, hipMemcpyHostToDevice);

  dim3 block(threads);
  int grid_dim_1d = (N / threads + 1);
  dim3 grid(grid_dim_1d);
  spdlog::trace("Grid dim (1D): {:d}", grid_dim_1d);
  spdlog::trace("Box size: {:d}", sizeof(Aabb));

  // int* rank;
  // hipMalloc((void**)&rank, sizeof(int)*(1*N));

  // int* rank_x = &rank[0];
  // int* rank_y = &rank[N];
  // int* rank_z = &rank[2*N];

  // Translate boxes -> SweepMarkers

  // hipEventRecord(start);
  // build_index<<<grid,block>>>(d_boxes, N, rank_x);
  // hipEventRecord(stop);
  // hipEventSynchronize(stop);

  // hipEventElapsedTime(&milliseconds, start, stop);

  // spdlog::trace("Elapsed time for build: {:.6f} ms", milliseconds);

  // Thrust sort (can be improved by sort_by_key)
  hipEventRecord(start);
  try {
    // thrust::sort_by_key(thrust::device, d_boxes, d_boxes + N, rank_x,
    // sort_aabb_x() );
    thrust::sort(thrust::device, d_boxes, d_boxes + N, sort_aabb_x());
  } catch (thrust::system_error &e) {
    spdlog::trace("Error: {:s} ", e.what());
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  spdlog::trace("Elapsed time for sort: {:.6f} ms", milliseconds);

  // Test print some sorted output
  // print_sort_axis<<<1,1>>>(d_boxes, 5);
  hipDeviceSynchronize();

  int devices_count;
  hipGetDeviceCount(&devices_count);
  // devices_count-=2;
  devices_count = devcount ? devcount : devices_count;
  int range = ceil((float)N / devices_count);

  // free(start);
  // free(stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipEvent_t starts[devices_count];
  hipEvent_t stops[devices_count];
  float millisecondss[devices_count];

  tbb::parallel_for(0, devices_count, 1, [&](int &device_id) {
    auto &local_overlaps = storages.local();

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    spdlog::trace("{:s} -> unifiedAddressing = {:d}", prop.name,
                  prop.unifiedAddressing);

    hipSetDevice(device_id);

    // hipEvent_t start, stop;
    hipEventCreate(&starts[device_id]);
    hipEventCreate(&stops[device_id]);

    int is_able;

    hipDeviceCanAccessPeer(&is_able, device_id, device_init_id);
    hipDeviceSynchronize();
    if (is_able) {
      hipDeviceEnablePeerAccess(device_init_id, 0);
      hipDeviceSynchronize();
    } else if (device_init_id != device_id)
      spdlog::trace("Device {:d} cant access Device {:d}", device_id,
                    device_init_id);

    int range_start = range * device_id;
    int range_end = range * (device_id + 1);
    spdlog::trace("device_id: {:d} [{:d}, {:d})", device_id, range_start,
                  range_end);

    Aabb *d_b;
    hipMalloc((void **)&d_b, sizeof(Aabb) * N);
    hipMemcpy(d_b, d_boxes, sizeof(Aabb) * N, hipMemcpyDefault);
    hipDeviceSynchronize();

    hipDeviceCanAccessPeer(&is_able, device_id, device_init_id);
    hipDeviceSynchronize();
    if (is_able) {
      hipDeviceDisablePeerAccess(device_init_id);
      hipDeviceSynchronize();
    } else if (device_init_id != device_id)
      spdlog::trace("Device {:d} cant access Device {:d}", device_id,
                    device_init_id);

    // Allocate counter to GPU + set to 0 collisions
    int *d_count;
    gpuErrchk(hipMalloc((void **)&d_count, sizeof(int)));
    gpuErrchk(hipMemset(d_count, 0, sizeof(int)));
    gpuErrchk(hipGetLastError());

    // Find overlapping pairs
    int guess = N * 200;
    spdlog::trace("Guess {:d}", guess);

    int2 *d_overlaps;
    hipMalloc((void **)&d_overlaps, sizeof(int2) * (guess));
    gpuErrchk(hipGetLastError());

    int grid_dim_1d = (range / threads + 1);
    dim3 grid(grid_dim_1d);

    int count;
    hipEventRecord(starts[device_id]);
    retrieve_collision_pairs<<<grid, block, smemSize>>>(
      d_b, d_count, d_overlaps, N, guess, nbox, range_start, range_end);
    hipEventRecord(stops[device_id]);
    hipEventSynchronize(stops[device_id]);
    hipEventElapsedTime(&millisecondss[device_id], starts[device_id],
                         stops[device_id]);
    hipDeviceSynchronize();
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    spdlog::trace("count for device {:d} : {:d}", device_id, count);

    if (count > guess) {
      spdlog::trace("Running again");
      hipFree(d_overlaps);
      hipMalloc((void **)&d_overlaps, sizeof(int2) * (count));
      // hipMemset(d_overlaps, 0, sizeof(int2)*(count));
      hipMemset(d_count, 0, sizeof(int));
      hipEventRecord(starts[device_id]);
      retrieve_collision_pairs<<<grid, block, smemSize>>>(
        d_b, d_count, d_overlaps, N, count, nbox, range_start, range_end);
      hipEventRecord(stops[device_id]);
      hipEventSynchronize(stops[device_id]);
      hipEventElapsedTime(&millisecondss[device_id], starts[device_id],
                           stops[device_id]);
      hipDeviceSynchronize();
      hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
      spdlog::trace("count2 for device {:d} : {:d}", device_id, count);
    }

    // spdlog::trace("Elapsed time: {:.9f} ms/collision", milliseconds/count);
    // spdlog::trace("Boxes: {:d}", N);
    // spdlog::trace("Elapsed time: {:.9f} ms/box", milliseconds/N);

    // int2 * overlaps = new int2[count];
    int2 *overlaps = (int2 *)malloc(sizeof(int2) * count);
    gpuErrchk(hipMemcpy(overlaps, d_overlaps, sizeof(int2) * (count),
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipGetLastError());

    spdlog::trace("Final count for device {:d}:  {:d}", device_id, count);

    // local_overlaps.reserve(local_overlaps.size() + count);

    // auto is_face = [&](Aabb x){return x.vertexIds.z >= 0;};
    // auto is_edge = [&](Aabb x){return x.vertexIds.z < 0 && x.vertexIds.y >= 0
    // ;}; auto is_vertex = [&](Aabb x){return x.vertexIds.z < 0  &&
    // x.vertexIds.y < 0;};

    for (size_t i = 0; i < count; i++) {
      // local_overlaps.emplace_back(overlaps[i].x, overlaps[i].y);
      // finOverlaps.push_back();
      int aid = overlaps[i].x;
      int bid = overlaps[i].y;
      Aabb a = boxes[aid];
      Aabb b = boxes[bid];

      if (is_vertex(a) && is_face(b)) // vertex, face
        local_overlaps.emplace_back(aid, bid);
      else if (is_edge(a) && is_edge(b))
        local_overlaps.emplace_back(min(aid, bid), max(aid, bid));
      else if (is_face(a) && is_vertex(b))
        local_overlaps.emplace_back(bid, aid);
    }

    spdlog::trace("Total(filt.) overlaps for devid {:d}: {:d}", device_id,
                  local_overlaps.size());
    // delete [] overlaps;
    // free(overlaps);

    // // free(counter);
    // // free(counter);
    // hipFree(d_overlaps);
    // hipFree(d_count);
    // hipFree(d_b);
    // hipFree(d_r);
    // hipDeviceReset();
  }); // end tbb for loop

  merge_local_overlaps(storages, finOverlaps);

  float longest = 0;
  for (int i = 0; i < devices_count; i++) {
    for (int j = 0; j < devices_count; j++) {
      hipEventElapsedTime(&milliseconds, starts[i], stops[j]);
      longest = milliseconds > longest ? milliseconds : longest;
    }
  }
  printf("\n");
  spdlog::trace("Elapsed time: {:.6f} ms", longest);
  spdlog::trace("Merged overlaps: {:d}", finOverlaps.size());
  printf("\n");
}

void run_sweep_sharedqueue(const Aabb *boxes, MemHandler *memhandle, int N,
                           int nbox,
                           std::vector<std::pair<int, int>> &finOverlaps,
                           int2 *&d_overlaps, int *&d_count, int &threads,
                           int &tidstart, int &devcount, const int memlimit) {
  hipDeviceSynchronize();
  spdlog::trace("Number of boxes: {:d}", N);

  if (!memhandle->MAX_OVERLAP_CUTOFF)
    memhandle->MAX_OVERLAP_CUTOFF = N;
  if (memlimit) {
    memhandle->limitGB = memlimit;
    spdlog::trace("Limit set to {:d}", memhandle->limitGB);
  }

  int device_init_id = 0;

  int smemSize;
  setup(device_init_id, smemSize, threads, nbox);

  hipSetDevice(device_init_id);

  // Allocate boxes to GPU
  Aabb *d_boxes;
  hipMalloc((void **)&d_boxes, sizeof(Aabb) * N);
  hipMemcpy(d_boxes, boxes, sizeof(Aabb) * N, hipMemcpyHostToDevice);

  int grid_dim_1d = (N / threads + 1);
  spdlog::trace("Grid dim (1D): {:d}", grid_dim_1d);
  spdlog::trace("Box size: {:d}", sizeof(Aabb));
  spdlog::trace("Scalar3 size: {:d}", sizeof(Scalar3));
  spdlog::trace("sizeof(queue) size: {:d}", sizeof(Queue));

  Scalar2 *d_sm;
  hipMalloc((void **)&d_sm, sizeof(Scalar2) * N);

  MiniBox *d_mini;
  hipMalloc((void **)&d_mini, sizeof(MiniBox) * N);

  // mean of all box points (used to find best axis)
  //   Scalar3 *d_mean;
  //   hipMalloc((void **)&d_mean, sizeof(Scalar3));
  //   hipMemset(d_mean, 0, sizeof(Scalar3));

  //   // recordLaunch("create_ds", grid_dim_1d, threads, smemSize, create_ds,
  //   // d_boxes, d_sm, d_mini, N, d_mean);
  //   recordLaunch("calc_mean", grid_dim_1d, threads, smemSize, calc_mean,
  //   d_boxes,
  //                d_mean, N);

  //   // temporary
  //   Scalar3 mean;
  //   hipMemcpy(&mean, d_mean, sizeof(Scalar3),
  //   hipMemcpyDeviceToHost); spdlog::trace("mean: x {:.6f} y {:.6f} z
  //   {:.6f}", mean.x, mean.y, mean.z);

  //   // calculate variance and determine which axis to sort on
  //   Scalar3 *d_var; // 2 vertices per box
  //   hipMalloc((void **)&d_var, sizeof(Scalar3));
  //   hipMemset(d_var, 0, sizeof(Scalar3));
  //   // calc_variance(boxes, d_var, N, d_mean);
  //   recordLaunch("calc_variance", grid_dim_1d, threads, smemSize,
  //   calc_variance,
  //                d_boxes, d_var, N, d_mean);
  //   hipDeviceSynchronize();

  //   Scalar3 var3d;
  //   hipMemcpy(&var3d, d_var, sizeof(Scalar3),
  //   hipMemcpyDeviceToHost); float maxVar = max(max(var3d.x, var3d.y),
  //   var3d.z);

  //   spdlog::trace("var: x {:.6f} y {:.6f} z {:.6f}", var3d.x, var3d.y,
  //   var3d.z);

  Dimension axis;
  //   if (maxVar == var3d.x)
  //     axis = x;
  //   else if (maxVar == var3d.y)
  //     axis = y;
  //   else
  //     axis = z;
  //   // hack
  axis = x;

  spdlog::trace("Axis: {:s}", axis == x ? "x" : (axis == y ? "y" : "z"));

  recordLaunch<Aabb *, Scalar2 *, MiniBox *, int, Dimension>(
    "create_ds", grid_dim_1d, threads, smemSize, create_ds, d_boxes, d_sm,
    d_mini, N, axis);

  try {
    thrust::sort_by_key(thrust::device, d_sm, d_sm + N, d_mini, sort_aabb_x());
  } catch (thrust::system_error &e) {
    spdlog::trace("Thrust error: {:s} ", e.what());
  }
  spdlog::trace("Thrust sort finished");

  gpuErrchk(hipGetLastError());

  // MemHandler memhandle;
  // Guessing global collision output size
  // int guess = memhandle->MAX_OVERLAP_CUTOFF; // 200 * N;
  spdlog::trace("Guess cutoff: {:d}", memhandle->MAX_OVERLAP_CUTOFF);
  size_t overlaps_size = memhandle->MAX_OVERLAP_SIZE * sizeof(int2);
  spdlog::trace("overlaps_size: {:d}", overlaps_size);
  gpuErrchk(hipGetLastError());

  int *d_start;
  int *d_end;
  // int boxes_done = tidstart;

  gpuErrchk(hipMalloc((void **)&d_start, sizeof(int)));
  gpuErrchk(hipMalloc((void **)&d_end, sizeof(int)));
  gpuErrchk(
    hipMemcpy(d_start, &tidstart, sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemset(d_end, 0, sizeof(int)));
  gpuErrchk(hipGetLastError());

  // int * d_count;
  gpuErrchk(hipMalloc((void **)&d_count, sizeof(int)));
  gpuErrchk(hipMemset(d_count, 0, sizeof(int)));

  // Device memhandler to keep track of vars
  MemHandler *d_memhandle;
  gpuErrchk(hipMalloc((void **)&d_memhandle, sizeof(MemHandler)));
  hipMemcpy(d_memhandle, memhandle, sizeof(MemHandler),
             hipMemcpyHostToDevice);

  // int2 * d_overlaps;
  spdlog::trace("Allocating overlaps memory");
  gpuErrchk(hipMalloc((void **)&d_overlaps, overlaps_size));

  spdlog::trace("Starting two stage_queue");
  spdlog::trace("Starting tid {:d}", tidstart);
  recordLaunch<Scalar2 *, const MiniBox *, int2 *, int, int *, int *, int *,
               MemHandler *>("twostage_queue_1st", grid_dim_1d, threads,
                             twostage_queue, d_sm, d_mini, d_overlaps, N,
                             d_count, d_start, d_end, d_memhandle);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipGetLastError());

  int count;
  gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
  spdlog::debug("1st count for device {:d}:  {:d}", device_init_id, count);

  int realcount;
  gpuErrchk(hipMemcpy(&realcount, &(d_memhandle->realcount), sizeof(int),
                       hipMemcpyDeviceToHost));
  spdlog::trace("Real count for device {:d}:  {:d}", device_init_id, realcount);

  // int diff = boxes_done;
  // gpuErrchk(
  //   hipMemcpy(&boxes_done, d_end, sizeof(int), hipMemcpyDeviceToHost));
  // diff = boxes_done - diff;

  spdlog::debug("realcount: {:d}, overlap_size {:d} -> Batching", realcount,
                memhandle->MAX_OVERLAP_SIZE);
  while (count > memhandle->MAX_OVERLAP_SIZE) {
    gpuErrchk(hipFree(d_overlaps));

    memhandle->handleBroadPhaseOverflow(count);

    gpuErrchk(hipMalloc((void **)&d_overlaps,
                         sizeof(int2) * (memhandle->MAX_OVERLAP_SIZE)));

    gpuErrchk(hipMemset(d_count, 0, sizeof(int)));
    gpuErrchk(hipMemset(d_end, 0, sizeof(int)));

    hipMemcpy(d_memhandle, memhandle, sizeof(MemHandler),
               hipMemcpyHostToDevice);

    recordLaunch<Scalar2 *, const MiniBox *, int2 *, int, int *, int *, int *,
                 MemHandler *>("twostage_queue_1st", grid_dim_1d, threads,
                               twostage_queue, d_sm, d_mini, d_overlaps, N,
                               d_count, d_start, d_end, d_memhandle);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    // gpuErrchk(hipMemcpy(&count, &(d_memhandle->realcount), sizeof(int),
    //                      hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&realcount, &(d_memhandle->realcount), sizeof(int),
                         hipMemcpyDeviceToHost));
    spdlog::trace("Real count for loop:  {:d}", realcount);
    spdlog::trace("Count for loop:  {:d}", count);
    // gpuErrchk(
    //   hipMemcpy(&boxes_done, d_end, sizeof(int), hipMemcpyDeviceToHost));
    spdlog::debug("Count {:d}, max size {:d}", realcount,
                  memhandle->MAX_OVERLAP_SIZE);
  }
  // tidstart = boxes_done;
  tidstart += memhandle->MAX_OVERLAP_CUTOFF;
  // spdlog::trace("Next threadid start {:d}", tidstart);

  gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
  // spdlog::trace("Final count for device {:d}:  {:d}", device_init_id, count);
  gpuErrchk(hipMemcpy(d_count, &(d_memhandle->realcount), sizeof(int),
                       hipMemcpyDeviceToDevice));
  // spdlog::trace("Final count for device {:d}:  {:d}", device_init_id, count);

  hipFree(d_boxes);
  hipFree(d_mini);
  hipFree(d_sm);
  hipFree(d_start);
  hipFree(d_end);
  hipFree(d_memhandle);

#ifdef KEEP_CPU_OVERLAPS
  int2 *overlaps = (int2 *)malloc(sizeof(int2) * count);
  gpuErrchk(hipMemcpy(overlaps, d_overlaps, sizeof(int2) * (count),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipGetLastError());

  spdlog::trace("Final count for device {:d}:  {:d}", 0, count);

  finOverlaps.reserve(finOverlaps.size() + count);
  for (int i = 0; i < count; i++) {
    finOverlaps.emplace_back(overlaps[i].x, overlaps[i].y);
  }

  free(overlaps);

  spdlog::trace("Total(filt.) overlaps for devid {:d}: {:d}", 0,
                finOverlaps.size());
#endif
  spdlog::trace("Next threadstart {:d}", tidstart);
}
} // namespace stq::gpu