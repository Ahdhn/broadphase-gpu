#include "hip/hip_runtime.h"
#include <gpubf/simulation.h>

#define BLOCK_SIZE_1D 32 //sqrt(MAX_BLOCK_SIZE)
#define MAX_BLOCK_SIZE 1024 //for 1080Ti, V100

void run_collision_counter(Aabb* boxes, int N) {
    // int N = 200000;
    // Aabb boxes[N];
    // for (int i = 0; i<N; i++)
    // {
    //     boxes[i] = Aabb(i);
    //     // printf("box %i created\n", boxes[i].id);
    // }

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    reset_counter<<<1,1>>>(d_counter);
    hipDeviceSynchronize();

     int collisions;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    // int bytes_mem_intrfce = 352 >> 3;
    // int mem_clock_rate = 1376 << 1;
    // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10, 3);

    // Set up timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get number of collisions
    hipEventRecord(start);
    count_collisions<<<1,1>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("(count_collisions<<<1,1>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    printf("Effective Bandwidth (GB/s): %.6f (GB/s)\n", 32*2/milliseconds/1e6);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<1,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<1,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<2,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<2,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<56,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<56,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<256,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<256,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    return;
    // printf("%zu\n", sizeof(Aabb));


    // Retrieve count from GPU and print out
    // int counter;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    // printf("count: %d\n", counter);
    // return 0;
}

void run_scaling(Aabb* boxes, int N, vector<unsigned long>& finOverlaps)
{
    // guess overlaps size
    int guess = 2*18*N;

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    reset_counter<<<1,1>>>(d_count);
    hipDeviceSynchronize();

    int * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int)*(guess));

    dim3 block(BLOCK_SIZE_1D,BLOCK_SIZE_1D);
    dim3 grid ( (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D,  (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D );

    get_collision_pairs<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, guess);
    hipDeviceSynchronize();

    int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    if (2*count > guess) //we went over
    {
        printf("Running again\n");
        hipFree(d_overlaps);
        hipMalloc((void**)&d_overlaps, sizeof(int)*(count) * 2);
        reset_counter<<<1,1>>>(d_count);
        hipDeviceSynchronize();
        get_collision_pairs<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, 2*count);
        hipDeviceSynchronize();
    }

    int * overlaps =  (int*)malloc(sizeof(int) * (count)*2);
    hipMemcpy( overlaps, d_overlaps, sizeof(int)*(count)*2, hipMemcpyDeviceToHost);


    hipFree(d_overlaps);
    for (size_t i=0; i< count; i++)
    {
        const Aabb& a = boxes[overlaps[2*i]];
        const Aabb& b = boxes[overlaps[2*i + 1]];
        if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
        {
            finOverlaps.push_back(a.ref_id);
            finOverlaps.push_back(b.ref_id);
        }
        else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
        {
            finOverlaps.push_back(b.ref_id);
            finOverlaps.push_back(a.ref_id);
        }
        else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
        {
            
            finOverlaps.push_back(min(a.ref_id, b.ref_id));
            finOverlaps.push_back(max(a.ref_id, b.ref_id));
        }
    }

    printf("Total overlaps: %lu\n", finOverlaps.size() / 2);
    free(overlaps);
    // free(counter);
    // free(counter);
    hipFree(d_count); 

}