#include "hip/hip_runtime.h"
#include <gpubf/simulation.h>

// #define MAX_CONST_MEM 65536
// #define MAX_CONSTANT_BOXES MAX_CONST_MEM / sizeof(Aabb)

int setup_shared_memory()
{
    // Host code
    int maxbytes = 98304; // 96 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(get_collision_pairs), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

    int smemSize;
    int devId = 0;
    hipDeviceGetAttribute(&smemSize, 
        hipDeviceAttributeSharedMemPerBlockOptin, devId);
    printf("Shared Memory per Block: %i B\n", smemSize);

    return smemSize;
}


void run_collision_counter(Aabb* boxes, int N) {

    // int N = 200000;
    // Aabb boxes[N];
    // for (int i = 0; i<N; i++)
    // {
    //     boxes[i] = Aabb(i);
    //     // printf("box %i created\n", boxes[i].id);
    // }

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    reset_counter<<<1,1>>>(d_counter);
    hipDeviceSynchronize();

     int collisions;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    // int bytes_mem_intrfce = 352 >> 3;
    // int mem_clock_rate = 1376 << 1;
    // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10, 3);

    // Set up timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get number of collisions
    hipEventRecord(start);
    count_collisions<<<1,1>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("(count_collisions<<<1,1>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    printf("Effective Bandwidth (GB/s): %.6f (GB/s)\n", 32*2/milliseconds/1e6);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<1,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<1,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<2,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<2,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<56,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<56,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<256,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<256,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    return;
    // printf("%zu\n", sizeof(Aabb));


    // Retrieve count from GPU and print out
    // int counter;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    // printf("count: %d\n", counter);
    // return 0;
}
// __constant__ Aabb d_boxes[MAX_CONSTANT_BOXES];
void run_scaling(const Aabb* boxes, int N, vector<unsigned long>& finOverlaps)
{
    int smemSize = setup_shared_memory();
    const int nBoxesPerThread = smemSize / sizeof(Aabb) / (2*BLOCK_SIZE_1D);
    printf("Boxes per Thread: %i\n", nBoxesPerThread);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // guess overlaps size
    int guess = 2*18*N;

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    reset_counter<<<1,1>>>(d_count);
    hipDeviceSynchronize();

    int * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int)*(guess));

    dim3 block(BLOCK_SIZE_1D,BLOCK_SIZE_1D);
    // dim3 grid ( (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D,  (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D );
    int grid_dim_1d = (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D / nBoxesPerThread;
    dim3 grid( grid_dim_1d, grid_dim_1d );

    hipEventRecord(start);
    get_collision_pairs<<<grid, block, nBoxesPerThread*2*BLOCK_SIZE_1D*sizeof(Aabb)>>>(d_boxes, d_count, d_overlaps, N, guess, nBoxesPerThread);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // hipDeviceSynchronize();

    int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (2*count > guess) //we went over
    {
        printf("Running again\n");
        hipFree(d_overlaps);
        hipMalloc((void**)&d_overlaps, sizeof(int)*(count) * 2);
        reset_counter<<<1,1>>>(d_count);
        hipDeviceSynchronize();
        hipEventRecord(start);
        get_collision_pairs<<<grid, block, nBoxesPerThread*2*BLOCK_SIZE_1D*sizeof(Aabb)>>>(d_boxes, d_count, d_overlaps, N, 2*count, nBoxesPerThread);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        // hipDeviceSynchronize();
    }
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Collisions: %i\n", count);
    printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
    printf("Boxes: %i\n", N);
    printf("Elapsed time: %.9f ms/box\n", milliseconds/N);

    int * overlaps =  (int*)malloc(sizeof(int) * (count)*2);
    hipMemcpy( overlaps, d_overlaps, sizeof(int)*(count)*2, hipMemcpyDeviceToHost);


    hipFree(d_overlaps);
    for (size_t i=0; i< count; i++)
    {
        const Aabb& a = boxes[overlaps[2*i]];
        const Aabb& b = boxes[overlaps[2*i + 1]];
        if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
        {
            finOverlaps.push_back(a.ref_id);
            finOverlaps.push_back(b.ref_id);
        }
        else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
        {
            finOverlaps.push_back(b.ref_id);
            finOverlaps.push_back(a.ref_id);
        }
        else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
        {
            
            finOverlaps.push_back(b.ref_id);
            finOverlaps.push_back(a.ref_id);
        }
    }

    printf("Total(filt.) overlaps: %lu\n", finOverlaps.size() / 2);
    free(overlaps);
    // free(counter);
    // free(counter);
    hipFree(d_count); 

}