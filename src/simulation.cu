#include "hip/hip_runtime.h"
#include <gpubf/simulation.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <tbb/mutex.h>
#include <tbb/parallel_for.h>
#include <tbb/blocked_range.h>
#include <tbb/task_scheduler_init.h>
#include <tbb/enumerable_thread_specific.h>
#include "tbb/concurrent_vector.h"

#include <cmath>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void setup(int devId, int& smemSize, int& threads, int& nboxes);



void run_collision_counter(Aabb* boxes, int N) {

    // int N = 200000;
    // Aabb boxes[N];
    // for (int i = 0; i<N; i++)
    // {
    //     boxes[i] = Aabb(i);
    //     // printf("box %i created\n", boxes[i].id);
    // }

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    reset_counter<<<1,1>>>(d_counter);
    hipDeviceSynchronize();

     int collisions;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    // int bytes_mem_intrfce = 352 >> 3;
    // int mem_clock_rate = 1376 << 1;
    // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10, 3);

    // Set up timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get number of collisions
    hipEventRecord(start);
    count_collisions<<<1,1>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("(count_collisions<<<1,1>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    printf("Effective Bandwidth (GB/s): %.6f (GB/s)\n", 32*2/milliseconds/1e6);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<1,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<1,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<2,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<2,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<56,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<56,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<256,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<256,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    return;
    // printf("%zu\n", sizeof(Aabb));


    // Retrieve count from GPU and print out
    // int counter;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    // printf("count: %d\n", counter);
    // return 0;
}

void run_scaling(const Aabb* boxes,  int N, int desiredBoxesPerThread, vector<unsigned long>& finOverlaps)
{

    int devId = 0;
    hipSetDevice(devId);

    int smemSize;
    int threads;

    setup(devId, smemSize, threads, desiredBoxesPerThread);
    const int nBoxesPerThread = desiredBoxesPerThread ? desiredBoxesPerThread : smemSize / sizeof(Aabb) / (2*(BLOCK_PADDED));
    printf("Boxes per Thread: %i\n", nBoxesPerThread);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // guess overlaps size
    int guess = 0;

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    reset_counter<<<1,1>>>(d_count);
    hipDeviceSynchronize();

    //Count collisions
    count_collisions<<<1,1>>>(d_boxes, d_count, N); 
    int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    reset_counter<<<1,1>>>(d_count);
    printf("Total collisions from counting: %i\n", count);



    int2 * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int2)*(guess));

    dim3 block(BLOCK_SIZE_1D,BLOCK_SIZE_1D);
    // dim3 grid ( (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D,  (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D );
    int grid_dim_1d = (N+BLOCK_SIZE_1D)/ BLOCK_SIZE_1D / nBoxesPerThread;
    dim3 grid( grid_dim_1d, grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));

    long long * d_queries;
    hipMalloc((void**)&d_queries, sizeof(long long)*(1));
    reset_counter<<<1,1>>>(d_queries);

    printf("Shared mem alloc: %i B\n", nBoxesPerThread*2*(BLOCK_PADDED)*sizeof(Aabb) );
    hipEventRecord(start);
    get_collision_pairs<<<grid, block, nBoxesPerThread*2*(BLOCK_PADDED)*sizeof(Aabb)>>>(d_boxes, d_count, d_overlaps, N, guess, nBoxesPerThread, d_queries);
    // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, guess);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // hipDeviceSynchronize();

    long long queries;
    hipMemcpy(&queries, d_queries, sizeof(long long), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("queries: %llu\n", queries);
    printf("needed queries: %llu\n", (long long)N*(N-1)/2 );

    // int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    

    if (count > guess) //we went over
    {
        printf("Running again\n");
        hipFree(d_overlaps);
        hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
        reset_counter<<<1,1>>>(d_count);
        hipDeviceSynchronize();
        hipEventRecord(start);
        get_collision_pairs<<<grid, block, nBoxesPerThread*2*(BLOCK_PADDED)*sizeof(Aabb)>>>(d_boxes, d_count, d_overlaps, N, count, nBoxesPerThread, d_queries);
        // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, 2*count);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        // hipDeviceSynchronize();
    }

    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Collisions: %i\n", count);
    printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
    printf("Boxes: %i\n", N);
    printf("Elapsed time: %.9f ms/box\n", milliseconds/N);
    // printf("Elapsed time: %.15f us/query\n", (milliseconds*1000)/((long long)N*N/2));

    int2 * overlaps =  (int2*)malloc(sizeof(int2) * (count));
    gpuErrchk(hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost));


    hipFree(d_overlaps);
    for (size_t i=0; i< count; i++)
    {
        // finOverlaps.push_back(overlaps[i].x);
        // finOverlaps.push_back(overlaps[i].y);
        
        const Aabb& a = boxes[overlaps[i].x];
        const Aabb& b = boxes[overlaps[i].y];
        if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
        {
            finOverlaps.push_back(a.ref_id);
            finOverlaps.push_back(b.ref_id);
        }
        else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
        {
            finOverlaps.push_back(b.ref_id);
            finOverlaps.push_back(a.ref_id);
        }
        else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
        {   
            finOverlaps.push_back(min(a.ref_id, b.ref_id));
            finOverlaps.push_back(max(a.ref_id, b.ref_id));
        }
    }

    printf("Total(filt.) overlaps: %lu\n", finOverlaps.size() / 2);
    free(overlaps);
    // free(counter);
    // free(counter);
    hipFree(d_count);
    hipDeviceReset();

}
//  // // //////// / / // / / // / // // / //  /

struct sort_sweepmarker_x
{
  __host__ __device__
  bool operator()(const SweepMarker &a, const SweepMarker &b) const {
    return (a.x < b.x);}
};

struct sort_aabb_x
{
  __host__ __device__
  bool operator()(const Aabb &a, const Aabb &b) const {
    return (a.min.x < b.min.x);}
};


void run_sweep(const Aabb* boxes, int N, int nbox, vector<pair<int,int>>& finOverlaps, int& threads)
{
    int devId = 0;
    hipSetDevice(devId);

    int smemSize;

    setup(devId, smemSize, threads, nbox);


    // int d_nbox;
    // hipMalloc((void**)&d_nbox, sizeof(int));
    // hipMemcpy(d_nbox, &nbox, sizeof(int), hipMemcpyHostToDevice);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    reset_counter<<<1,1>>>(d_count);
    hipDeviceSynchronize();


    // int SWEEP_BLOCK_SIZE = 1024;
    
    // maxBlockSize = 512;
    dim3 block(threads);
    int grid_dim_1d = (N / threads + 1); 
    dim3 grid( grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));
    printf("SweepMarker size: %i\n", sizeof(SweepMarker));

    // int* d_index;
    // hipMalloc((void**)&d_index, sizeof(int)*(N));
    int* rank;
    hipMalloc((void**)&rank, sizeof(int)*(1*N));

    int* rank_x = &rank[0];
    // int* rank_y = &rank[N];
    // int* rank_z = &rank[2*N];

    // Translate boxes -> SweepMarkers
    hipEventRecord(start);
    build_index<<<grid,block>>>(d_boxes, N, rank_x);
    // build_index<<<grid,block>>>(d_boxes, N, rank_y);
    // build_index<<<grid,block>>>(d_boxes, N, rank_z);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time for build: %.6f ms\n", milliseconds);

    // Thrust sort (can be improved by sort_by_key)
    hipEventRecord(start);
    // thrust::sort(thrust::device, d_axis, d_axis + N, sort_sweepmarker_x() );
    try{
        thrust::sort_by_key(thrust::device, d_boxes, d_boxes + N, rank_x, sort_aabb_x() );
        }
    catch (thrust::system_error &e){
        printf("Error: %s \n",e.what());}
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time for sort: %.6f ms\n", milliseconds);

    // Test print some sorted output
    // print_sort_axis<<<1,1>>>(d_boxes,rank_x, 5);
    hipDeviceSynchronize();

    // Find overlapping pairs
    int guess = 0;
    int2 * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int2)*(guess));

    int count;
    retrieve_collision_pairs<<<grid, block, smemSize>>>(d_boxes, rank_x, d_count, d_overlaps, N, guess, nbox);
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (count > guess) //we went over
    {
        printf("Running again\n");
        hipFree(d_overlaps);
        hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
        reset_counter<<<1,1>>>(d_count);
        hipDeviceSynchronize();
        hipEventRecord(start);
        retrieve_collision_pairs<<<grid, block, smemSize>>>(d_boxes, rank_x, d_count, d_overlaps, N, count, nbox);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Elapsed time for findoverlaps: %.6f ms\n", milliseconds);
    }
    // int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Collisions: %i\n", count);
    printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
    printf("Boxes: %i\n", N);
    printf("Elapsed time: %.9f ms/box\n", milliseconds/N);

    int2 * overlaps =  (int2*)malloc(sizeof(int2) * (count));
    hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost);

    printf("Final count: %i\n", count);

    hipFree(d_overlaps);
    for (size_t i=0; i < count; i++)
    {
        // finOverlaps.push_back(overlaps[i].x);
        // finOverlaps.push_back(overlaps[i].y);
        
        const Aabb& a = boxes[overlaps[i].x];
        const Aabb& b = boxes[overlaps[i].y];
        if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
        {
            finOverlaps.emplace_back(a.ref_id, b.ref_id);
        }
        else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
        {
            finOverlaps.emplace_back(b.ref_id, a.ref_id);
        }
        else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
        {
            finOverlaps.emplace_back(min(a.ref_id, b.ref_id), max(a.ref_id, b.ref_id));
        }
    }

    printf("Total(filt.) overlaps: %lu\n", finOverlaps.size() );
    free(overlaps);
    // free(counter);
    // free(counter);
    hipFree(d_count); 

    hipDeviceReset();
}

// 
// 
// 
// MULTI GPU SWEEP SUPPORT
void merge_local_overlaps(
    const tbb::enumerable_thread_specific<tbb::concurrent_vector<std::pair<int,int>>>& storages,
    std::vector<std::pair<int,int>>& overlaps)
{
    overlaps.clear();
    size_t num_overlaps = overlaps.size();
    for (const auto& local_overlaps : storages) {
        num_overlaps += local_overlaps.size();
    }
    // serial merge!
    overlaps.reserve(num_overlaps);
    for (const auto& local_overlaps : storages) {
        overlaps.insert(
            overlaps.end(), local_overlaps.begin(), local_overlaps.end());
    }
}

void run_sweep_multigpu(const Aabb* boxes, int N, int nbox, vector<pair<int, int>>& finOverlaps, int& threads)
{
    cout<<"default threads "<<tbb::task_scheduler_init::default_num_threads()<<endl;
    // tbb::task_scheduler_init init(2);
    tbb::enumerable_thread_specific<tbb::concurrent_vector<pair<int,int>>> storages;
    // tbb::enumerable_thread_specific<vector<int2>> storages2;


    float milliseconds = 0;
    int device_init_id = 0;

    int smemSize;
    setup(device_init_id, smemSize, threads, nbox);

    hipSetDevice(device_init_id);

    // int d_nbox;
    // hipMalloc((void**)&d_nbox, sizeof(int));
    // hipMemcpy(d_nbox, &nbox, sizeof(int), hipMemcpyHostToDevice);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    dim3 block(threads);
    int grid_dim_1d = (N / threads + 1); 
    dim3 grid( grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));

    int* rank;
    hipMalloc((void**)&rank, sizeof(int)*(1*N));

    int* rank_x = &rank[0];
    // int* rank_y = &rank[N];
    // int* rank_z = &rank[2*N];

    // Translate boxes -> SweepMarkers

    hipEventRecord(start);
    build_index<<<grid,block>>>(d_boxes, N, rank_x);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time for build: %.6f ms\n", milliseconds);

    // Thrust sort (can be improved by sort_by_key)
    hipEventRecord(start);
    // thrust::sort(thrust::device, d_axis, d_axis + N, sort_sweepmarker_x() );
    try{
        thrust::sort_by_key(thrust::device, d_boxes, d_boxes + N, rank_x, sort_aabb_x() );
        }
    catch (thrust::system_error &e){
        printf("Error: %s \n",e.what());}
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time for sort: %.6f ms\n", milliseconds);


    // Test print some sorted output
    // print_sort_axis<<<1,1>>>(d_boxes,rank_x, 5);
    hipDeviceSynchronize();
    

    int devices_count;
    hipGetDeviceCount(&devices_count);
    // devices_count--;
    int range = ceil( N / devices_count); 

    tbb::parallel_for(0, devices_count, 1, [&](int & device_id)    {
        
        hipSetDevice(device_id);
        int is_able;

        for (int i=0; i<devices_count; i++)
        {
            hipDeviceCanAccessPeer(&is_able, device_id, i);
            if (is_able)
            { 
                hipDeviceEnablePeerAccess(i, 0);  
            }
            else if (i != device_id)
                printf("Device %i cant access Device %i\n", device_id, i);
        }
        

        // gpuErrchk( hipGetLastError() );   
        // gpuErrchk( hipGetLastError() );
        // int canAccessPeer = 0;
        // hipDeviceCanAccessPeer(&accessPair, device_id, device_init_id);
        // if (canAccessPeer)
    
        // hipMemcpyPeerAsync ( void* dst, int  dstDevice, const void* src, int  srcDevice, size_t count, hipStream_t stream = 0 )


        int range_start  = range*device_id;
        int range_end = range*(device_id + 1);
        printf("device_id: %i [%i, %i)\n", device_id, range_start, range_end);
        

        // Aabb * d_b;
        // hipMalloc((void**)&d_b, sizeof(Aabb)*N);
        // if (device_id == device_init_id )
        //     hipMemcpy(d_b, d_boxes, sizeof(Aabb)*N, hipMemcpyDeviceToDevice);
        // else
        //     hipMemcpyPeer ( d_b, device_id, d_boxes, device_init_id, sizeof(Aabb)*N);

        // int * d_r;
        // hipMalloc((void**)&d_r, sizeof(int)*(1*N));
        // if (device_id == device_init_id )
        //     hipMemcpy(d_r, rank, sizeof(int)*N, hipMemcpyDeviceToDevice);
        // else
        //     hipMemcpyPeer( d_r, device_id, rank, device_init_id, sizeof(int)*N);
        

        
        // Allocate counter to GPU + set to 0 collisions
        int * d_count;
        hipMalloc((void**)&d_count, sizeof(int));
        reset_counter<<<1,1>>>(d_count);
        hipDeviceSynchronize();

        // Find overlapping pairs
        int guess = 0;
        int2 * d_overlaps;
        hipMalloc((void**)&d_overlaps, sizeof(int2)*(guess));

        int count;
        retrieve_collision_pairs<<<grid, block, smemSize>>>(d_boxes, rank_x, d_count, d_overlaps, N, guess, nbox, range_start, range_end);
        hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("Count for 1st run %i for device %i\n", count, device_id);

        if (count > guess) //we went over
        {
            printf("Running again\n");
            hipFree(d_overlaps);
            hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
            reset_counter<<<1,1>>>(d_count);
            hipDeviceSynchronize();
            hipEventRecord(start);

            retrieve_collision_pairs<<<grid, block, smemSize>>>(d_boxes, rank_x, d_count, d_overlaps, N, count, nbox, range_start, range_end);
            // gpuErrchk( hipGetLastError() );
            // gpuErrchk( hipDeviceSynchronize() );
            
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
        }
        // int count;
        hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        
        printf("Collisions: %i\n", count);
        printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
        printf("Boxes: %i\n", N);
        printf("Elapsed time: %.9f ms/box\n", milliseconds/N);

        int2 * overlaps =  (int2*)malloc(sizeof(int2) * (count));
        // auto& local_overlaps2 = storages2.local();
        // local_overlaps2.resize(count);
        hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        printf("Final count for device %i:  %i\n", device_id, count);
        printf("overlaps.x for dev: %i %i\n", device_id, overlaps[0].x);

        
        auto& local_overlaps = storages.local();
        local_overlaps.reserve(local_overlaps.size() + count);
        
        for (size_t i=0; i < count; i++)
        {
            // finOverlaps.push_back(overlaps[i].x);
            // finOverlaps.push_back(overlaps[i].y);
            
            Aabb a = boxes[overlaps[i].x];
            Aabb b = boxes[overlaps[i].y];
            
        
            if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
            {
                local_overlaps.emplace_back(a.ref_id, b.ref_id);
            }
            else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
            {
                local_overlaps.emplace_back(b.ref_id, a.ref_id);
            }
            else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
            {
                local_overlaps.emplace_back(min(a.ref_id, b.ref_id), max(a.ref_id, b.ref_id));
            }
        }
        
        printf("Total(filt.) overlaps for devid %i: %i\n", device_id, local_overlaps.size());
        // free(overlaps);
        
        // // free(counter);
        // // free(counter);
        hipFree(d_overlaps);
        hipFree(d_count); 
        // hipFree(d_b);
        // hipFree(d_r);
        hipDeviceReset();

    }); //end tbb for loop

    merge_local_overlaps(storages, finOverlaps);

    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Merged overlaps: %i\n", finOverlaps.size());

}