// goal is to understand global/shared memory access and bank conflicts

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reverse(int* d_nums, int *d_rev, int N)
{
    extern __shared__ int s_nums[];
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

    if (tid >= N) return;

    for (int i = 0; i < 1; i++)
    {
        s_nums[N-1-tid] = d_nums[tid];
    }

    for (int i = 0; i < 1; i++)
    {
        d_rev[tid] = s_nums[tid];
    }

}


int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    const int N = 20;
    int nums[N];
    int rev[N];
        
    for (int i = 0; i<N; i++)
    {
        nums[i] = i;
        rev[i] = 0;
    }
    
    int * d_nums;
    int * d_rev; 
    hipMalloc((void**)&d_nums, sizeof(int)*N);
    hipMemcpy(d_nums, nums, sizeof(int)*N, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_rev, sizeof(int)*N);

    int ITER = 100;
    float milliseconds = 0;
    float avg_ms = 0;
    for (int i=0; i < ITER; i++)
    {
    hipEventRecord(start);
    reverse<<<1,N, sizeof(int)*N>>>(d_nums, d_rev, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    avg_ms += milliseconds;
    }
    avg_ms /= ITER;
    printf("Avg. elapsed time: %.6f ms\n", avg_ms);

    hipMemcpy(rev, d_rev, sizeof(int)*N, hipMemcpyDeviceToHost);
    for (int i=0; i < N; i++)
        printf("%i ", rev[i]);
    printf("\n");
}

