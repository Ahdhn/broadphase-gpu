#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <bitset>
#include <string>
#include <cuda/pipeline>
#include <cuda/semaphore>
#include <hip/hip_cooperative_groups.h>

#include <gpubf/queue.cuh>
#include <gpubf/aabb.cuh>

using namespace std;
typedef long long int ll;

__global__ void run(ll* in, ll * out, int N)
{
    __shared__ cuda::pipeline_shared_state<cuda::thread_scope_block, 1> pss;
    __shared__ Queue queue;
    queue.capacity = HEAP_SIZE;

    // extern __shared__ T s[];
    auto group = cooperative_groups::this_thread_block();
    // T* shared[2] = { s, s + 2 * group.size() };

      // Create a partitioned block-scoped pipeline where half the threads are producers.
    cuda::std::size_t producer_count = group.size() / 2;
    cuda::pipeline<cuda::thread_scope_block> pipe = cuda::make_pipeline(group, &pss, producer_count);

    // cuda::std::ptrdiff_t max = 1;
    extern __shared__ cuda::binary_semaphore<cuda::thread_scope_block> a[];
    cuda::binary_semaphore<cuda::thread_scope_block>* b[1] = {a};
    // a = cuda::binary_semaphore<cuda::thread_scope_block>(1);
    a[0].release();
    __syncthreads();

    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid >= N) return;

    // Prime the pipeline.
    // pipe.producer_acquire();
    int2 val = make_int2(in[tid], in[tid]);
    if (a[0].try_acquire())
    {
        printf("tid %i acquired semaphore\n", tid);
        queue.push(val);
    }
    else
        printf("tid %i failed to acquire semaphore\n", tid);
    

    // pipe.producer_commit();

    // cuda::pipeline_consumer_wait_prior<1>(pipe);
    // pipe.consumer_wait();
    // // while (queue.size())
    // int2 res = queue.pop();
    // out[tid] = val.x * val.y;
    // pipe.consumer_release();
    // // Create a pipeline.

    // out[tid] = // atomicAdd(&var[0].x, __powf(boxes[tid].min.x-mean[0].x, 2));
    // out[tid] = __mulhi(f1,f2);
    
    return;

}

int main( int argc, char **argv )
{
    vector<ll> nums;

    int N = atoi(argv[1]);


    for (ll i = 0; i < N; i++)
    {
        nums.push_back(i);
    }

    ll * d_in;
    hipMalloc((void**)&d_in, sizeof(ll)*N);
    hipMemcpy(d_in, nums.data(), sizeof(ll)*N, hipMemcpyHostToDevice);

    ll * d_out;
    hipMalloc((void**)&d_out, sizeof(ll)*N);
    hipMemset(d_out, 0, sizeof(ll)*N);

    int block = 1024;
    int grid = (N / block + 1); 
    printf("grid size: %i\n", grid);

    run<<<grid, block, 80>>>(d_in, d_out, N);
    hipDeviceSynchronize();

    vector<ll> out;
    out.reserve(N);
    hipMemcpy(out.data(), d_out, sizeof(ll)*N, hipMemcpyDeviceToHost);


    for (ll i = 0; i < N; i++)
    {        
        printf("%lld:%lld ", nums[i], out[i]);
    }
    printf("\n");

}