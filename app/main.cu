#include "hip/hip_runtime.h"
#include <assert.h>
#include <ctype.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// #define CCD_USE_DOUBLE

#include <stq/gpu/groundtruth.cuh>
#include <stq/gpu/simulation.cuh>
#include <stq/gpu/util.cuh>
#include <stq/gpu/memory.cuh>
// #include <stq/gpu/klee.cuh>
#include <stq/gpu/io.cuh>

#include <spdlog/spdlog.h>

using namespace std;
using namespace stq::gpu;

// spdlog::set_level(spdlog::level::trace);
#ifdef _WIN32
char *optarg = NULL;
int optind = 1;
int getopt(int argc, char *const argv[], const char *optstring) {
  if ((optind >= argc) || (argv[optind][0] != '-') || (argv[optind][0] == 0)) {
    return -1;
  }

  int opt = argv[optind][1];
  const char *p = strchr(optstring, opt);

  if (p == NULL) {
    return '?';
  }
  if (p[1] == ':') {
    optind++;
    if (optind >= argc) {
      return '?';
    }
    optarg = argv[optind];
    optind++;
  }
  return opt;
}
#else
#include <unistd.h>
#endif



bool is_file_exist(const char *fileName) {
  ifstream infile(fileName);
  return infile.good();
}

int main(int argc, char **argv) {
  spdlog::set_level(static_cast<spdlog::level::level_enum>(0));
  vector<char *> compare;

  MemHandler *memhandle = new MemHandler();

  char *filet0;
  char *filet1;

  filet0 = argv[1];
  if (is_file_exist(argv[2]))
    filet1 = argv[2];
  else
    filet1 = argv[1];

  vector<Aabb> boxes;
  Eigen::MatrixXd vertices_t0;
  Eigen::MatrixXd vertices_t1;
  Eigen::MatrixXi faces;
  Eigen::MatrixXi edges;

  parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
  constructBoxes(vertices_t0, vertices_t1, edges, faces, boxes);
  size_t N = boxes.size();

  int nbox = 0;
  int parallel = 0;
  bool evenworkload = false;
  int devcount = 1;
  bool pairing = false;
  bool sharedqueue_mgpu = false;
  bool bigworkerqueue = false;

  int memlimit = 0;

  int o;
  while ((o = getopt(argc, argv, "c:n:b:p:d:v:WPQZ")) != -1) {
    switch (o) {
    case 'c':
      optind--;
      for (; optind < argc && *argv[optind] != '-'; optind++) {
        compare.push_back(argv[optind]);
        // compare_mathematica(overlaps, argv[optind]);
      }
      break;
    case 'n':
      N = atoi(optarg);
      break;
    case 'b':
      nbox = atoi(optarg);
      break;
    case 'v':
      memlimit = atoi(optarg);
      break;
    case 'p':
      parallel = stoi(optarg);
      break;
    case 'd':
      devcount = atoi(optarg);
      break;
    case 'W':
      evenworkload = true;
      break;
    case 'P':
      pairing = true;
      break;
    case 'Q':
      sharedqueue_mgpu = true;
      break;
    case 'Z':
      bigworkerqueue = true;
      break;
    }
  }

  vector<pair<int, int>> overlaps;
  int2 *d_overlaps; // device
  int *d_count;     // device
  int tidstart = 0;

  if (evenworkload)
    run_sweep_sharedqueue(boxes.data(), memhandle, N, nbox, overlaps,
                          d_overlaps, d_count, parallel, tidstart, devcount,
                          memlimit);
  // else if (sharedqueue_mgpu)
  //   run_sweep_multigpu_queue(boxes.data(), N, nbox, overlaps, parallel,
  //                            devcount);
  // else if (bigworkerqueue)
  //   run_sweep_bigworkerqueue(boxes.data(), N, nbox, overlaps, d_overlaps,
  //                            d_count, parallel, devcount);
  else
    run_sweep_multigpu(boxes.data(), N, nbox, overlaps, parallel, devcount);

  spdlog::debug("Final CPU overlaps size : {:d}", overlaps.size());

  for (auto i : compare) {
    compare_mathematica(overlaps, i);
  }
}
