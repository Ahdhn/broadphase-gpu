#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <igl/readOBJ.h>
#include <igl/readPLY.h>
#include <igl/edges.h>

#include <gpubf/simulation.h>
#include <gpubf/groundtruth.h>
#include <gpubf/util.cuh>
#include <gpubf/klee.cuh>
#include <gpubf/io.cuh>

using namespace std;

int main( int argc, char **argv )
{
    vector<char*> compare;

    const char* filet0 = argv[1];
    const char* filet1 = argv[2];
    
    vector<Aabb> boxes;
    parseMesh(filet0, filet1, boxes);
    int N = boxes.size();
    int nbox = 0;
    
    int o;
    while ((o = getopt (argc, argv, "c:n:b:")) != -1)
    {
        switch (o)
        {
            case 'c':
                optind--;
                for( ;optind < argc && *argv[optind] != '-'; optind++)
                {
                    compare.push_back(argv[optind]);
                    // compare_mathematica(overlaps, argv[optind]); 
                }
                break;
            case 'n':
                N = atoi(optarg);
                break;
            case 'b':
                nbox = atoi(optarg);
                break;
        }
    }

    vector<unsigned long> overlaps;
    // int i = 1;
    // while (i < N)
    // {
    //     run_scaling(boxes.data(), i, overlaps);
    //     printf("\n");
    //     i = i << 1;
    // }
    run_sweep(boxes.data(), N, nbox, overlaps);
    for (auto i : compare)
    {
        // printf("%s\n", i );
        compare_mathematica(overlaps, i);
    }
    cout << endl;

}