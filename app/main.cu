#include "hip/hip_runtime.h"
#include <assert.h>
#include <ctype.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// #define CCD_USE_DOUBLE

#include <gpubf/groundtruth.cuh>
#include <gpubf/simulation.cuh>
#include <gpubf/util.cuh>
// #include <gpubf/klee.cuh>
#include <gpubf/io.cuh>

#include <spdlog/spdlog.h>

using namespace std;
using namespace ccd::gpu;

// spdlog::set_level(spdlog::level::trace);

bool is_file_exist(const char *fileName) {
  ifstream infile(fileName);
  return infile.good();
}

int main(int argc, char **argv) {
  spdlog::set_level(static_cast<spdlog::level::level_enum>(0));
  vector<char *> compare;

  char *filet0;
  char *filet1;

  filet0 = argv[1];
  if (is_file_exist(argv[2]))
    filet1 = argv[2];
  else
    filet1 = argv[1];

  vector<Aabb> boxes;
  Eigen::MatrixXd vertices_t0;
  Eigen::MatrixXd vertices_t1;
  Eigen::MatrixXi faces;
  Eigen::MatrixXi edges;

  parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
  constructBoxes(vertices_t0, vertices_t1, edges, faces, boxes);
  size_t N = boxes.size();

  int nbox = 0;
  int parallel = 0;
  bool evenworkload = false;
  int devcount = 1;
  bool pairing = false;
  bool sharedqueue_mgpu = false;
  bool bigworkerqueue = false;

  int o;
  while ((o = getopt(argc, argv, "c:n:b:p:d:WPQZ")) != -1) {
    switch (o) {
    case 'c':
      optind--;
      for (; optind < argc && *argv[optind] != '-'; optind++) {
        compare.push_back(argv[optind]);
        // compare_mathematica(overlaps, argv[optind]);
      }
      break;
    case 'n':
      N = atoi(optarg);
      break;
    case 'b':
      nbox = atoi(optarg);
      break;
    case 'p':
      parallel = stoi(optarg);
      break;
    case 'd':
      devcount = atoi(optarg);
      break;
    case 'W':
      evenworkload = true;
      break;
    case 'P':
      pairing = true;
      break;
    case 'Q':
      sharedqueue_mgpu = true;
      break;
    case 'Z':
      bigworkerqueue = true;
      break;
    }
  }

  vector<pair<int, int>> overlaps;
  int2 *d_overlaps; // device
  int *d_count;     // device

  if (evenworkload)
    run_sweep_sharedqueue(boxes.data(), N, nbox, overlaps, d_overlaps, d_count,
                          parallel, devcount);
  else if (sharedqueue_mgpu)
    run_sweep_multigpu_queue(boxes.data(), N, nbox, overlaps, parallel,
                             devcount);
  else if (bigworkerqueue)
    run_sweep_bigworkerqueue(boxes.data(), N, nbox, overlaps, d_overlaps,
                             d_count, parallel, devcount);
  else
    run_sweep_multigpu(boxes.data(), N, nbox, overlaps, parallel, devcount);

  for (auto i : compare) {
    compare_mathematica(overlaps, i);
  }
}
