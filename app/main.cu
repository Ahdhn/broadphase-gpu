#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <igl/readOBJ.h>
#include <igl/edges.h>

#include <gpubf/aabb.h>

using namespace std;

__global__ void count_collisions(Aabb * boxes, int * count, int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;
    
    const Aabb& a = boxes[tid];
    const Aabb& b = boxes[tid];
    // Aabb b = boxes[tid];

    bool collides = 
        a.max[0] >= b.min[0] && a.min[0] <= b.max[0] &&
        a.max[1] >= b.min[1] && a.min[1] <= b.max[1] &&
        a.max[2] >= b.min[2] && a.min[2] <= b.max[2];

    if (collides)
        atomicAdd(count, 1);
}

__global__ void reset_counter(int * counter){
    *counter = 0;
}

int run_simulation(Aabb* boxes, int N) {
    // int N = 200000;
    // Aabb boxes[N];
    // for (int i = 0; i<N; i++)
    // {
    //     boxes[i] = Aabb(i);
    //     // printf("box %i created\n", boxes[i].id);
    // }

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    reset_counter<<<1,1>>>(d_counter);
    hipDeviceSynchronize();

     int collisions;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    // int bytes_mem_intrfce = 352 >> 3;
    // int mem_clock_rate = 1376 << 1;
    // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10, 3);

    // Set up timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get number of collisions
    hipEventRecord(start);
    count_collisions<<<1,1>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("(count_collisions<<<1,1>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    printf("Effective Bandwidth (GB/s): %.6f (GB/s)\n", 32*2/milliseconds/1e6);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<1,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<1,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<2,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<2,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<56,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<56,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<256,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<256,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    return 1;
    // printf("%zu\n", sizeof(Aabb));


    // Retrieve count from GPU and print out
    // int counter;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    // printf("count: %d\n", counter);
    // return 0;
}

void constructBoxes
(
    Eigen::MatrixXd& vertices_t0, 
    Eigen::MatrixXd& vertices_t1, 
    Eigen::MatrixXi& faces, 
    Eigen::MatrixXi& edges, 
    vector<Aabb>& boxes
)
{
   addEdges(vertices_t0, vertices_t1, edges, boxes);
   addVertices(vertices_t0, vertices_t1, boxes);
   addFaces(vertices_t0, vertices_t1, faces, boxes);
}

void parseMesh(const char* filet0, const char* filet1, vector<Aabb>& boxes)
{

    // read in vertices, faces t=0
    Eigen::MatrixXd V0;
    Eigen::MatrixXi F;
    igl::readOBJ(filet0, V0, F);

    // get edges and close file
    Eigen::MatrixXi E;
    igl::edges(F,E);

    // read in vertices, t=1
    // faces should be same F^{t=0} = F^{t=1}
    Eigen::MatrixXd V1;    
    igl::readOBJ(filet1, V1, F);

    constructBoxes(V0, V1, F, E, boxes);
}


int main( int argc, const char* argv[] )
{
    const char* filet0 = argv[argc-2];
    const char* filet1 = argv[argc-1];
    
    vector<Aabb> boxes;
    parseMesh(filet0, filet1, boxes);

    Aabb* boxes_ptr = boxes.data();

    run_simulation(boxes_ptr, boxes.size());
}