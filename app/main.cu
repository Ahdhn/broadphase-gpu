#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <igl/readOBJ.h>
#include <igl/readPLY.h>
#include <igl/edges.h>

#include <gpubf/simulation.h>
#include <gpubf/groundtruth.h>
#include <gpubf/util.cuh>
#include <gpubf/klee.cuh>
#include <gpubf/io.cuh>

using namespace std;

int main( int argc, char **argv )
{
    vector<char*> compare;

    const char* filet0 = argv[1];
    const char* filet1 = argv[2];
    
    vector<Aabb> boxes;
    Eigen::MatrixXd vertices_t0;
    Eigen::MatrixXd vertices_t1;
    Eigen::MatrixXi faces; 
    Eigen::MatrixXi edges;

    parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
    constructBoxes(vertices_t0, vertices_t1, faces, edges, boxes);
    int N = boxes.size();
    int nbox = 0;
    int parallel = 0;
    // bool distributed = false;
    int devcount = 1;

    int o;
    while ((o = getopt (argc, argv, "c:n:b:p:d:")) != -1)
    {
        switch (o)
        {
            case 'c':
                optind--;
                for( ;optind < argc && *argv[optind] != '-'; optind++)
                {
                    compare.push_back(argv[optind]);
                    // compare_mathematica(overlaps, argv[optind]); 
                }
                break;
            case 'n':
                N = atoi(optarg);
                break;
            case 'b':
                nbox = atoi(optarg);
                break;
            case 'p':
                parallel = stoi(optarg);
                break;
            case 'd':
                devcount = atoi(optarg);
                break;
            // case 'i':
            //     devcount = atoi(optarg);
            //     break;
        }
    }

    vector<pair<int,int>> overlaps;
    // if (distributed)
        run_sweep_multigpu(boxes.data(), N, nbox, overlaps, parallel, devcount);
    // else
    //     run_sweep(boxes.data(), N, nbox, overlaps, parallel);
    for (auto i : compare)
    {
        // printf("%s\n", i );
        compare_mathematica(overlaps, i);
    }
    cout << endl;

}
