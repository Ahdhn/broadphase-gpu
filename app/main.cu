#include "hip/hip_runtime.h"
#include <assert.h>
#include <ctype.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// #define CCD_USE_DOUBLE

#include <gpubf/groundtruth.cuh>
#include <gpubf/simulation.cuh>
#include <gpubf/util.cuh>
// #include <gpubf/klee.cuh>
#include <gpubf/io.cuh>

using namespace std;
using namespace ccdgpu;

bool is_file_exist(const char *fileName) {
  ifstream infile(fileName);
  return infile.good();
}

int main(int argc, char **argv) {
  vector<char *> compare;

  char *filet0;
  char *filet1;

  filet0 = argv[1];
  if (is_file_exist(argv[2]))
    filet1 = argv[2];
  else
    filet1 = argv[1];

  vector<Aabb> boxes;
  Eigen::MatrixXd vertices_t0;
  Eigen::MatrixXd vertices_t1;
  Eigen::MatrixXi faces;
  Eigen::MatrixXi edges;

  parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
  constructBoxes(vertices_t0, vertices_t1, faces, edges, boxes);
  size_t N = boxes.size();
  std::cout << boxes.size() << std::endl;
  printf("Boxes (N): %i\n", N);
  int nbox = 0;
  int parallel = 0;
  bool evenworkload = false;
  int devcount = 1;
  bool pairing = false;
  bool sharedqueue_mgpu = false;
  bool bigworkerqueue = false;

  int o;
  while ((o = getopt(argc, argv, "c:n:b:p:d:WPQZ")) != -1) {
    switch (o) {
    case 'c':
      optind--;
      for (; optind < argc && *argv[optind] != '-'; optind++) {
        compare.push_back(argv[optind]);
        // compare_mathematica(overlaps, argv[optind]);
      }
      break;
    case 'n':
      N = atoi(optarg);
      break;
    case 'b':
      nbox = atoi(optarg);
      break;
    case 'p':
      parallel = stoi(optarg);
      break;
    case 'd':
      devcount = atoi(optarg);
      break;
    case 'W':
      evenworkload = true;
      break;
    case 'P':
      pairing = true;
      break;
    case 'Q':
      sharedqueue_mgpu = true;
      break;
    case 'Z':
      bigworkerqueue = true;
      break;
    }
  }

  vector<pair<int, int>> overlaps;
  int2 *d_overlaps; // device
  int *d_count;     // device

  if (evenworkload)
    run_sweep_sharedqueue(boxes.data(), N, nbox, overlaps, d_overlaps, d_count,
                          parallel, devcount);
  //   else if (pairing)
  //     run_sweep_pairing(boxes.data(), N, nbox, overlaps, parallel, devcount);
  else if (sharedqueue_mgpu)
    run_sweep_multigpu_queue(boxes.data(), N, nbox, overlaps, parallel,
                             devcount);
  else if (bigworkerqueue)
    run_sweep_bigworkerqueue(boxes.data(), N, nbox, overlaps, d_overlaps,
                             d_count, parallel, devcount);
  else
    run_sweep_multigpu(boxes.data(), N, nbox, overlaps, parallel, devcount);

  for (auto i : compare) {
    // printf("%s\n", i );
    compare_mathematica(overlaps, i);
  }
  cout << endl;
}
