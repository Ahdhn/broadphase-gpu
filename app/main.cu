#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <igl/readOBJ.h>
#include <igl/readPLY.h>
#include <igl/edges.h>

#include <gpubf/simulation.h>
#include <gpubf/groundtruth.h>
#include <gpubf/util.cuh>
#include <gpubf/klee.cuh>
#include <gpubf/io.cuh>

using namespace std;

int main( int argc, char **argv )
{
    vector<char*> compare;

    const char* filet0 = argv[1];
    const char* filet1 = argv[2];
    
    vector<Aabb> boxes;
    Eigen::MatrixXd vertices_t0;
    Eigen::MatrixXd vertices_t1;
    Eigen::MatrixXi faces; 
    Eigen::MatrixXi edges;

    parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
    constructBoxes(vertices_t0, vertices_t1, faces, edges, boxes);
    int N = boxes.size();
    int nbox = 0;
    int parallel = 0;

    int o;
    while ((o = getopt (argc, argv, "c:n:b:p:")) != -1)
    {
        switch (o)
        {
            case 'c':
                optind--;
                for( ;optind < argc && *argv[optind] != '-'; optind++)
                {
                    compare.push_back(argv[optind]);
                    // compare_mathematica(overlaps, argv[optind]); 
                }
                break;
            case 'n':
                N = atoi(optarg);
                break;
            case 'b':
                nbox = atoi(optarg);
                break;
            case 'p':
                parallel = stoi(optarg);
                break;
        }
    }

    vector<unsigned long> overlaps;
    // int i = 1;
    // while (i < N)
    // {
    //     run_scaling(boxes.data(), i, overlaps);
    //     printf("\n");
    //     i = i << 1;
    // }
    run_sweep(boxes.data(), N, nbox, overlaps, parallel);
    for (auto i : compare)
    {
        // printf("%s\n", i );
        compare_mathematica(overlaps, i);
    }
    cout << endl;

}